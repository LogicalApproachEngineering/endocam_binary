/*
 * Copyright (c) 2016-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include <cairo.h>
#include <pango/pango.h>
#include <pango/pango-layout.h>
#include <pango/pangocairo.h>


#include "customer_functions.h"
#include "cudaEGL.h"
#include "iva_metadata.h"

#define BOX_W 32
#define BOX_H 32

#define CORD_X 64
#define CORD_Y 64
#define MAX_BUFFERS 30
// static BBOX rect_data[MAX_BUFFERS];

/**
  * Dummy custom pre-process API implematation.
  * It just access mapped surface userspace pointer &
  * memset with specific pattern modifying pixel-data in-place.
  *
  * @param sBaseAddr  : Mapped Surfaces pointers
  * @param smemsize   : surfaces size array
  * @param swidth     : surfaces width array
  * @param sheight    : surfaces height array
  * @param spitch     : surfaces pitch array
  * @param nsurfcount : surfaces count
  */
static void
pre_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{ 
  static cairo_surface_t *mask_surface = cairo_image_surface_create_from_png("Square_Mask_v2p00.png");
  static cairo_surface_t *surface;
  static cairo_t *cairo_context;
  if (sformat[0] == COLOR_FORMAT_RGBA) {
    // unsigned char xoffset = 100;
    // unsigned char yoffset = 100;
    // unsigned char* rgba = (unsigned char *)sBaseAddr[0];
    // rgba += ((spitch[0] * yoffset) + xoffset);
    // cairo_surface_t *surface, *mask_surface;
    // cairo_t *cairo_context;

    surface = cairo_image_surface_create_for_data
            ((unsigned char *)sBaseAddr[0], CAIRO_FORMAT_ARGB32, swidth[0],
             sheight[0], spitch[0]);
    cairo_context = cairo_create (surface);

    // mask_surface = cairo_image_surface_create_from_png("Square_Mask_v2p00.png");
    cairo_set_source_surface(cairo_context, mask_surface, 0.0, 0.0);
    cairo_paint(cairo_context);

    cairo_destroy (cairo_context);
    cairo_surface_destroy (surface);
  }
}

/**
  * Dummy custom post-process API implematation.
  * It just access mapped surface userspace pointer &
  * memset with specific pattern modifying pixel-data in-place.
  *
  * @param sBaseAddr  : Mapped Surfaces pointers
  * @param smemsize   : surfaces size array
  * @param swidth     : surfaces width array
  * @param sheight    : surfaces height array
  * @param spitch     : surfaces pitch array
  * @param nsurfcount : surfaces count
  */
static void
post_process (void **sBaseAddr,
                unsigned int *smemsize,
                unsigned int *swidth,
                unsigned int *sheight,
                unsigned int *spitch,
                ColorFormat  *sformat,
                unsigned int nsurfcount,
                void ** usrptr)
{
  static cairo_surface_t *mask_surface = cairo_image_surface_create_from_png("Round_Mask_v2p00.png");
  static cairo_surface_t *surface;
  static cairo_t *cairo_context;
  if (sformat[0] == COLOR_FORMAT_RGBA) {
    // unsigned char xoffset = 100;
    // unsigned char yoffset = 100;
    // unsigned char* rgba = (unsigned char *)sBaseAddr[0];
    // rgba += ((spitch[0] * yoffset) + xoffset);
    // cairo_surface_t *surface;
    // cairo_t *cairo_context;

    surface = cairo_image_surface_create_for_data
            ((unsigned char *)sBaseAddr[0], CAIRO_FORMAT_ARGB32, swidth[0],
             sheight[0], spitch[0]);
    cairo_context = cairo_create (surface);

    // mask_surface = cairo_image_surface_create_from_png("Round_Mask_v2p00.png");
    cairo_set_source_surface(cairo_context, mask_surface, 0.0, 0.0);
    cairo_paint(cairo_context);
    // layout = pango_cairo_create_layout (cairo_context);
    // snprintf(font_size_buffer, 256, "%s %d",
    //     "Arial", 18);
    // desc = pango_font_description_from_string (font_size_buffer);
    // pango_layout_set_font_description (layout, desc);
    // snprintf(display_text, 256, "%s",
    //     "Hello, world.");
    // pango_layout_set_text (layout, display_text, -1);
    // cairo_set_source_rgba (cairo_context, 1.0, 0, 1.0, 1.0);
    // pango_cairo_update_layout (cairo_context, layout);
    // cairo_move_to(cairo_context, 256, 256);
    // pango_cairo_show_layout (cairo_context, layout);
    // pango_font_description_free (desc);
    // g_object_unref(layout);

    cairo_destroy (cairo_context);
    cairo_surface_destroy (surface);
  }

}



static void add_metadata(void ** usrptr)
{}

/**
  * Performs CUDA Operations on egl image.
  *
  * @param image : EGL image
  */
static void
gpu_process (EGLImageKHR image, void ** usrptr)
{
  hipError_t status;
  CUeglFrame eglFrame;
  hipGraphicsResource_t pResource = NULL;

  hipFree(0);
  status = cuGraphicsEGLRegisterImage(&pResource, image, CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLRegisterImage failed : %d \n", status);
    return;
  }

  status = cuGraphicsResourceGetMappedEglFrame( &eglFrame, pResource, 0, 0);
  if (status != hipSuccess) {
    printf ("hipGraphicsSubResourceGetMappedArray failed\n");
  }

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed \n");
  }

  if (eglFrame.frameType == CU_EGL_FRAME_TYPE_PITCH) {
    if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_ABGR) {
    /* Rectangle label in plane RGBA, you can replace this with any cuda algorithms */
      //addLabels((hipDeviceptr_t) eglFrame.frame.pPitch[0], eglFrame.pitch);
    } else if (eglFrame.eglColorFormat == CU_EGL_COLOR_FORMAT_YUV420_SEMIPLANAR) {
    /* Rectangle label in plan UV , you can replace this with any cuda algorithms */
      //addLabels((hipDeviceptr_t) eglFrame.frame.pPitch[1], eglFrame.pitch);
    } else
      printf ("Invalid eglcolorformat\n");
  }

  add_metadata(usrptr);

  status = hipCtxSynchronize();
  if (status != hipSuccess) {
    printf ("hipCtxSynchronize failed after memcpy \n");
  }

  status = hipGraphicsUnregisterResource(pResource);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLUnRegisterResource failed: %d \n", status);
  }
}

extern "C" void
init (CustomerFunction * pFuncs)
{
  pFuncs->fPreProcess = pre_process;
  pFuncs->fGPUProcess = gpu_process;
  pFuncs->fPostProcess = post_process;
}

extern "C" void
deinit (void)
{
  /* deinitialization */
}
